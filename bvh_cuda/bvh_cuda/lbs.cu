#include "hip/hip_runtime.h"
#include "lbs.h"

__global__ void blendVertex(float* mesh, float3* vertices, float* handles, float* transformation, float* weight_map, int N, int M)
{
	int index = threadIdx.x + blockDim.x*blockIdx.x; //Calculate vertex index
	//printf("%d ", index);
	if (index >= N) return;

	float sum_weight = 0;
	for (int i = 0; i < M; i++)
	{
		sum_weight += weight_map[index*M + i];
	}
	float sum[4] = { 0 };

	for (int i = 0; i < M; i++)
	{
		float offset[4], coordinate[4] = { 0 };
		float L2W[4][4];

		offset[0] = mesh[index * 3 + 0] - handles[i * 3 + 0];
		offset[1] = mesh[index * 3 + 1] - handles[i * 3 + 1];
		offset[2] = mesh[index * 3 + 2] - handles[i * 3 + 2];
		offset[3] = 1;
		//transformation 4m*4

		for (int j = 0; j < 4; j++)
		{
			for (int k = 0; k < 4; k++)
			{
				L2W[j][k] = transformation[(4 * i + j) * 4 + k];
			}
		}

		for (int j = 0; j < 4; j++)
		{
			for (int k = 0; k < 4; k++)
			{
				coordinate[j] += L2W[j][k] * offset[k];
			}
		}

		sum[0] += coordinate[0] * weight_map[index*M + i] / sum_weight;
		sum[1] += coordinate[1] * weight_map[index*M + i] / sum_weight;
		sum[2] += coordinate[2] * weight_map[index*M + i] / sum_weight;
		sum[3] += coordinate[3] * weight_map[index*M + i] / sum_weight;
	}

	//vertices[index * 3 + 0] = sum[0];
	//vertices[index * 3 + 1] = sum[1];
	//vertices[index * 3 + 2] = sum[2];
	vertices[index] = { sum[0],sum[1],sum[2] };
}


void LBS(float* mesh_dev, float* handles_dev, float* transformation_dev, float* weight_map_dev,
	float* transformation_host, int N, int M, struct hipGraphicsResource*& cuda_vbo_resource)
{

	hipMemcpy(transformation_dev, transformation_host, 4 * M * 4 * sizeof(float), hipMemcpyHostToDevice);

	float3* dptr = NULL;
	size_t num_bytes;

	hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes, cuda_vbo_resource);

	dim3 block(128);
	dim3 grid((N - 1) / 128 + 1);

	blendVertex << <grid, block >> > (mesh_dev, dptr, handles_dev, transformation_dev, weight_map_dev, N, M);


	hipDeviceSynchronize();

	hipGraphicsUnmapResources(1, &cuda_vbo_resource,0);

}

void mallocSpaceInCuda(float*& mesh_host,  float*& handles_host, float*& transformation_host, float*& weight_map_host,
	float*& mesh_dev, float*& handles_dev, float*& transformation_dev, float*& weight_map_dev,
	int N, int M)
{

	mesh_dev = NULL;
	handles_dev = NULL;
	transformation_dev = NULL;
	weight_map_dev = NULL;

	hipMalloc((void**)&mesh_dev, N * 3 * sizeof(float));
	hipMalloc((void**)&handles_dev, M * 3 * sizeof(float));
	hipMalloc((void**)&transformation_dev, 4 * M * 4 * sizeof(float));
	hipMalloc((void**)&weight_map_dev, N*M * sizeof(float));
	hipMemcpy(mesh_dev, mesh_host, N * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(handles_dev, handles_host, M * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(weight_map_dev, weight_map_host, N*M * sizeof(float), hipMemcpyHostToDevice);
}

void freeSpaceInCuda(float* mesh_host, float* handles_host, float* transformation_host, float* weight_map_host,
	float* mesh_dev, float* handles_dev, float* transformation_dev, float* weight_map_dev,
	int N, int M)
{
	hipFree(mesh_dev);
	hipFree(handles_dev);
	hipFree(transformation_dev);
	hipFree(weight_map_dev);

	free(mesh_host);
	free(handles_host);
	free(transformation_host);
	free(weight_map_host);

	hipDeviceReset();
}
